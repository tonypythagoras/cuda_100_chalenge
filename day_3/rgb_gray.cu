#include "hip/hip_runtime.h"

#include  <cstdio>
#include  <iostream>



// the kernel that does the actual of mixing red, green and blue, turning the mixture to gray.
__global__ void rgb_to_gray(float *red, float *green, float *blue , float *gray, int width, int height){

   unsigned int row=blockIdx.y * blockDim.y+threadIdx.y;
   unsigned int col= blockIdx.x * blockDim.x + threadIdx.x;
   int globalId =row * width + col;
   if(row < height && col < width){
      
      gray[globalId]=red[globalId]*3/10 + green[globalId]*6/10 + blue[globalId]*1/10;

   }


}




int main(){

// the size of the color array
const unsigned int N=256;
const int size = N*sizeof(int);

const int width=16;
const int height=16;

// allocate memories in the host for the clor arrays
float *red =new float[N];
float *green = new float[N];
float *blue =new float[N];
float *gray= new float[N];

// initialize the colors
for(int i=0;i<N; i++){
       red[i]=255;
       green[i]=128;
       blue[i]=255;
}

float *red_d;
float *blue_d;
float *green_d;
float *gray_d;

// allocate memory in the device
hipMalloc((void**)&red_d,size);
hipMalloc((void**)&green_d,size);
hipMalloc((void**)&blue_d,size);
hipMalloc((void**)&gray_d,size);

// copy memory to device
hipMemcpy(red_d,red,size,hipMemcpyHostToDevice);
hipMemcpy(blue_d,blue,size,hipMemcpyHostToDevice);
hipMemcpy(green_d,green,size,hipMemcpyHostToDevice);

// allocate the threads and the thread blocks
dim3 numThreadPerBlock(8,8);
dim3 numberOfBlocks((width + numThreadPerBlock.x -1)/numThreadPerBlock.x,(height + numThreadPerBlock.y -1)/numThreadPerBlock.x);

// calls the kernel tha does the job of mixing the colors of red, green and blue, turning them to gray
  rgb_to_gray <<< numberOfBlocks, numThreadPerBlock >>>(red_d, green_d, blue_d, gray_d, width, height);

hipMemcpy(gray,gray_d,size,hipMemcpyDeviceToHost);


// prints out the final output which is the gray
for(int i=0;i < N; i++){
    std::cout<<gray[i]<<std::endl;
}

// free allocated device memory
hipFree(red_d);
hipFree(blue_d);
hipFree(green_d);
hipFree(gray_d);

// free allocated host memory
delete[] red;
delete[] blue;
delete[] green;
delete[] gray;


return 0;






}
