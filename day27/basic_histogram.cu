#include "hip/hip_runtime.h"

#include "iostream"
#include <stdlib.h>
#include <stdio.h>


unsigned const int N = 16;
 
 // This computes the frequency of numbers in an array
__global__ void hisKernel(int *numbers, int *frequency){

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride= blockDim.x * gridDim.x;
    while(i < N){
      atomicAdd(&(frequency[numbers[i]]),1);
      i=i+stride;
    }



}


// init array
void initArray(int *numbers){

numbers[0]=9;
numbers[1]=9;
numbers[2]=8;
numbers[3]=8;

numbers[4]=1;
numbers[5]=1;
numbers[6]=1;
numbers[7]=1;

numbers[8]=3;
numbers[9]=3;
numbers[10]=3;
numbers[11]=3;
numbers[12]=3;

numbers[13]=1;
numbers[14]=1;
numbers[15]=8;

    // for(int i=0;i< N; i++){
    //    if((i/10)==0){
    //    numbers[i]=10;
    //    }else if((i/50)==0){
    //     numbers[i]=50;
    //    }else if((i/100)==0){
    //     numbers[i]=100;
    //    }else if((i/200)==0){
    //     numbers[i]=200;
    //    }else if((i/250)==0){
    //     numbers[i]=250;
    //    }else if((i/300)==0){
    //     numbers[i]=300;
    //    }else if((i/400)==0){
    //     numbers[i]=400;
    //    }else{
    //     numbers[i]=i;
    //    }

   // }
}





int main(){

size_t size = N * sizeof(int);
// allocates some host memories
int *numbers =(int*)malloc(size);
int *frequency =(int*)malloc(size);

initArray(numbers);

int *numbers_d;
int *frequency_d;


//allocates some device memories
hipError_t err;
err = hipMalloc((void**)&numbers_d,size);

if(err !=hipSuccess){
     fprintf(stderr, "hipMalloc d_A error: %s\n", hipGetErrorString(err)); exit(EXIT_FAILURE); 
}

err = hipMalloc((void**)&frequency_d,size);

if(err !=hipSuccess){
     fprintf(stderr, "hipMalloc d_A error: %s\n", hipGetErrorString(err)); exit(EXIT_FAILURE); 
}
//copy memory from host to the device
hipMemcpy(numbers_d,numbers,size,hipMemcpyHostToDevice);


// initializes thread and threadblocks
dim3 threadPerBlock(8);
dim3 numberOfBlocks(2);

// kernel launch
hisKernel <<<numberOfBlocks, threadPerBlock >>> (numbers_d, frequency_d);

//copty memory from device to host
hipMemcpy(frequency,frequency_d,size,hipMemcpyDeviceToHost);
//display result
for (int i=0;i<N;i++){
    std::cout<< frequency[i]<< std::endl;
   }


//de-allocate device memories
hipFree(frequency_d);
hipFree(numbers_d);


//de-allocates host memories
delete[] frequency;
delete[] numbers;

    return 0;
}
