#include "hip/hip_runtime.h"

#include "iostream"
#include <stdio.h>
#include <stdlib.h>

unsigned const int N=8;
unsigned const int P = 4;
unsigned const int NUM_BLOCK=4;
unsigned const int THREAD_PER_BLOCK=4;


struct COO {  
  int len;           
  int *row;
  int *col;
  int *values;
};




// This performs a sparse matrix multiplication using coo(cordinate format)
__global__ void cooSparseMatrixKernel(COO *coo, int *invector, int *outvector){
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   int toAdd=0;
   if(i < (coo->len)){
    int row = coo->row[i];
    int col = coo->col[i];
    int value = coo->values[i];
     toAdd=invector[col]*value;
     atomicAdd(&outvector[row], toAdd);
   }
      

}






// initializes data
void initArray(int *row, int *col, int *values, int *vect){
  values[0]=1;
  values[1]=7;
  values[2]=5;
  values[3]=3;
  values[4]=9;
  values[5]=2;
  values[6]=8;
  values[7]=6;

  col[0]=0;
  col[1]=1;
  col[2]=0;
  col[3]=2;
  col[4]=3;
  col[5]=1;
  col[6]=2;
  col[7]=3;


  row[0]=0;
  row[1]=0;
  row[2]=1;
  row[3]=1;
  row[4]=1;
  row[5]=2;
  row[6]=2;
  row[7]=3;

  vect[0]=2;
  vect[1]=1;
  vect[2]=3;
  vect[3]=1;





    
}


int main(){
size_t size= N * sizeof(int);
size_t size_p= P * sizeof(int);
size_t coo_size= (3 * size) + 1;

int *invector_h;
int *outvector_h;
int *row_h;
int *col_h;
int *values_h;
// allocate memories on the host
row_h=(int*)malloc(size);
col_h=(int*)malloc(size);
values_h=(int*)malloc(size);
invector_h=(int*)malloc(size_p);
outvector_h=(int*)malloc(size_p);



initArray(row_h, col_h, values_h,invector_h);
     COO myCOO;


     myCOO.len=8;
     myCOO.row=row_h;
     myCOO.col=col_h;
     myCOO.values=values_h;

     COO *myCOO_D;




int *invector_d;
int *outvector_d;
int *row_d;
int *col_d;
int *values_d;




hipError_t err;
// allocates memories on the device

err=hipMalloc((void**)&myCOO_D, sizeof(COO));
 if(err !=hipSuccess){
               fprintf(stderr, "hipMalloc d_A error: %s\n", hipGetErrorString(err)); exit(EXIT_FAILURE); 
       }


err=hipMalloc((void**)&row_d,size);
 if(err !=hipSuccess){
               fprintf(stderr, "hipMalloc d_A error: %s\n", hipGetErrorString(err)); exit(EXIT_FAILURE); 
       }


err=hipMalloc((void**)&row_d,size);
 if(err !=hipSuccess){
               fprintf(stderr, "hipMalloc d_A error: %s\n", hipGetErrorString(err)); exit(EXIT_FAILURE); 
       }

err=hipMalloc((void**)&col_d,size);
 if(err !=hipSuccess){
               fprintf(stderr, "hipMalloc d_A error: %s\n", hipGetErrorString(err)); exit(EXIT_FAILURE); 
       }
err = hipMalloc((void**)&values_d,size);
 if(err !=hipSuccess){
               fprintf(stderr, "hipMalloc d_A error: %s\n", hipGetErrorString(err)); exit(EXIT_FAILURE); 
       }

       err = hipMalloc((void**)&invector_d,size_p);
 if(err !=hipSuccess){
               fprintf(stderr, "hipMalloc d_A error: %s\n", hipGetErrorString(err)); exit(EXIT_FAILURE); 
       }

err = hipMalloc((void**)&outvector_d,size_p);
 if(err !=hipSuccess){
               fprintf(stderr, "hipMalloc d_A error: %s\n", hipGetErrorString(err)); exit(EXIT_FAILURE); 
       }


//copy memory from host to device
    hipMemcpy(myCOO_D,&myCOO,sizeof(COO),hipMemcpyHostToDevice);

    hipMemcpy(values_d,values_h,size,hipMemcpyHostToDevice);
    hipMemcpy(col_d,col_h,size,hipMemcpyHostToDevice);
    hipMemcpy(row_d,row_h,size,hipMemcpyHostToDevice);

    hipMemcpy(invector_d,invector_h,size_p,hipMemcpyHostToDevice);

     


     
     // inilizes thread and threadblocks
     dim3 threadPerBlock(8);
     dim3 numberOfBlocks(1);

    // launches kernel for execution
     cooSparseMatrixKernel <<< numberOfBlocks, threadPerBlock >>> (myCOO_D, invector_d, outvector_d);

    // copy memories fro device to host
     hipMemcpy(outvector_h, outvector_d,size_p,hipMemcpyDeviceToHost);
         

   // output display
 for (int i=0;i<P;i++){
    std::cout<< outvector_h[i]<< std::endl;
   }

// free some device memories
hipFree(myCOO_D);
hipFree(outvector_d);
hipFree(invector_d);
hipFree(col_d);
hipFree(row_d);
hipFree(values_d);



// free some host memories
delete[] outvector_h;
delete[] invector_h;
delete[] col_h;
delete[] row_h;
delete[] values_h;



    return 0;
}