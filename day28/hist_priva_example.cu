#include "hip/hip_runtime.h"

#include "iostream"
#include <stdlib.h>
#include <stdio.h>


//constant
unsigned const int N = 16;

// This kernel calculates the frequency of numbers inside an array using privatization technique
__global__ void hisKernel(int *numbers, int *frequency){
    
    __shared__ float sh[N];
    sh[threadIdx.x]=0;
    __syncthreads();

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride= blockDim.x * gridDim.x;
   
    while(i < N){
      atomicAdd(&(sh[numbers[i]]),1);
      i=i+stride;
    }
    __syncthreads();

    if(threadIdx.x < N){
    atomicAdd(&(frequency[threadIdx.x]),sh[threadIdx.x]);
    }


}


// init the array with numbers
void initArray(int *numbers){

numbers[0]=9;
numbers[1]=9;
numbers[2]=8;
numbers[3]=8;

numbers[4]=1;
numbers[5]=1;
numbers[6]=1;
numbers[7]=1;

numbers[8]=3;
numbers[9]=3;
numbers[10]=3;
numbers[11]=3;
numbers[12]=3;

numbers[13]=1;
numbers[14]=1;
numbers[15]=8;
}





int main(){
// calculates the size of the array
size_t size = N * sizeof(int);

// allocate memory on the host
int *numbers =(int*)malloc(size);
int *frequency =(int*)malloc(size);

initArray(numbers);

int *numbers_d;
int *frequency_d;

// allocate memory on the device
hipError_t err;
err = hipMalloc((void**)&numbers_d,size);

if(err !=hipSuccess){
     fprintf(stderr, "hipMalloc d_A error: %s\n", hipGetErrorString(err)); exit(EXIT_FAILURE); 
}

err = hipMalloc((void**)&frequency_d,size);

if(err !=hipSuccess){
     fprintf(stderr, "hipMalloc d_A error: %s\n", hipGetErrorString(err)); exit(EXIT_FAILURE); 
}

// copy memory to the device
hipMemcpy(numbers_d,numbers,size,hipMemcpyHostToDevice);

// initailizes thread and threadblocks
dim3 threadPerBlock(16);
dim3 numberOfBlocks(1);

// launches the kernel
hisKernel <<<numberOfBlocks, threadPerBlock >>> (numbers_d, frequency_d);

// copy memory from device to host
hipMemcpy(frequency,frequency_d,size,hipMemcpyDeviceToHost);

// display the result
for (int i=0;i<N;i++){
    std::cout<< frequency[i]<< std::endl;
   }

// de-allocate device memories
hipFree(frequency_d);
hipFree(numbers_d);


//de-allocates host memories
delete[] frequency;
delete[] numbers;

    return 0;
}
