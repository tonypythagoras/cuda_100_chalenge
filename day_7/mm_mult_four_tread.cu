#include "hip/hip_runtime.h"

#include <cstdio>
#include <iostream>


//this is the kernel that does the matrix multiplication but with only one thread
__global__ void mm_four_tread(int *a, int *b, int *c, int width,int N){

    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
     int globalId=row*width+col;


if(globalId < N){
    // assigs the row multiplicant
     int j=globalId % width;
     // checks the row multiplicant
     int myRow=globalId/width;
     int sum=0;
     for(int i=0;i < width; i++){
             sum +=a[myRow*width+i]*b[j];
             j +=width;

     }
     c[globalId]=sum;
}

}


int main(){

    unsigned int N= 256;
    unsigned int size= N*sizeof(int);
    unsigned int width= 16;

    int *a=new int[N];
    int *b= new int[N];
    int *c = new int[N];


    // initialize the array in the host
     for (int i=0;i<N; i++){
       a[i]=i+2;
       b[i]=i+1;
     }

     int *a_d;
     int *b_d;
     int *c_d;
     
     // allocate memories on the device
     hipMalloc((void**)&a_d,size);
     hipMalloc((void**)&b_d,size);
     hipMalloc((void**)&c_d,size);


// copy memories from host to device
     hipMemcpy(a_d,a,size,hipMemcpyHostToDevice);
     hipMemcpy(b_d,b,size,hipMemcpyHostToDevice);

      // allocate thread and thread blocks
     dim3 threadPerBlock(1,2);
     dim3 numberOfBlocks(threadPerBlock.x, threadPerBlock.y);



//kernel call
mm_four_tread <<<numberOfBlocks, threadPerBlock >>>(a_d,b_d,c_d,width, N);
// copy memory from device to host
 hipMemcpy(c,c_d,size,hipMemcpyDeviceToHost);


// print out the result in the the host
 for(int i=0;i<N;i++){
    std::cout<<c[i]<<std::endl;
 }

// free allocated device memories
hipFree(a_d);
hipFree(b_d);
hipFree(c_d);

// free allocated host memories
delete[] a;
delete[] b;
delete[] c;

return 0;









}