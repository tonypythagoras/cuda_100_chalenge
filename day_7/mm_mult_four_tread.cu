#include "hip/hip_runtime.h"

#include <cstdio>
#include <iostream>


//this is the kernel that does the matrix multiplication but with only one thread
__global__ void mm_four_tread(float *a, float *b, float *c, int width,int N){

    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
     int globalId=row*width+col;


if(globalId < N){
    // assigs the row multiplicant
     int j=globalId % width;
     // checks the row multiplicant
     int myRow=globalId/width;
     int sum=0;
     for(int i=0;i < width; i++){
             sum +=a[myRow*width+i]*b[j];
             j +=width;

     }
     c[globalId]=sum;
}

}


int main(){

    unsigned int N= 16;
    unsigned int size= N*sizeof(float);
    unsigned int width= 4;

    float *a=new float[N];
    float *b= new float[N];
    float *c = new float[N];


    // initialize the array in the host
     for (int i=0;i<N; i++){
      if(i==4||i==13||i==15){
       a[i]=2.0;
      }else{
         a[i]=1.0;
      }
       b[i]=2.0;
     }


     float *a_d;
     float *b_d;
     float *c_d;
     
     // allocate memories on the device
     hipMalloc((void**)&a_d,size);
     hipMalloc((void**)&b_d,size);
     hipMalloc((void**)&c_d,size);


// copy memories from host to device
     hipMemcpy(a_d,a,size,hipMemcpyHostToDevice);
     hipMemcpy(b_d,b,size,hipMemcpyHostToDevice);

      // allocate thread and thread blocks
     dim3 threadPerBlock(2,2);
     dim3 numberOfBlocks((width + threadPerBlock.x -1)/threadPerBlock.x, (width + threadPerBlock.y - 1)/threadPerBlock.y);



//kernel call
mm_four_tread <<<numberOfBlocks, threadPerBlock >>>(a_d,b_d,c_d,width, N);
// copy memory from device to host
 hipMemcpy(c,c_d,size,hipMemcpyDeviceToHost);


// print out the result in the the host
 for(int i=0;i<N;i++){
    std::cout<<c[i]<<std::endl;
 }

// free allocated device memories
hipFree(a_d);
hipFree(b_d);
hipFree(c_d);

// free allocated host memories
delete[] a;
delete[] b;
delete[] c;

return 0;

}