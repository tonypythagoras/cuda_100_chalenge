#include "hip/hip_runtime.h"

#include "iostream"
#include <stdio.h>
#include <stdlib.h>

unsigned const int N=16;
unsigned const int P = 4;
unsigned const int NUM_BLOCK=4;
unsigned const int THREAD_PER_BLOCK=4;


// This performs an exclusive scan of numbers in an array
__global__ void scanKernel(float *input, float *vector, float * partial_sum){
   int i = blockIdx.x * blockDim.x + threadIdx.x;


   if(i == 0){
     vector[i]= 0;}
   else{
     vector[i]= input[i-1];
   }
     __syncthreads();

     for(int stride=1; stride <= THREAD_PER_BLOCK/2; stride *=2){
         float temp;
          if(threadIdx.x >= stride){
            temp=vector[i-stride];            
          }
          __syncthreads();
          if(threadIdx.x >= stride){
            vector[i] +=temp;
          }
          __syncthreads();
     }
     if(threadIdx.x == blockDim.x-1){
     partial_sum[blockIdx.x]=vector[i];
     }

}



__global__ void scanPartial(float * partial_sum,float * partial_sum_t){
   int i = blockIdx.x * blockDim.x + threadIdx.x;
     if(i < (blockDim.x)){

      if(i == 0){
     partial_sum_t[i]= 0;}
   else{
     partial_sum_t[i]= partial_sum[i-1];
   }
     __syncthreads();



     for(int stride=1; stride <= (blockDim.x/2); stride *=2){

            float temp;
          if(threadIdx.x >= stride){
            temp=partial_sum_t[i-stride];            
          }
          __syncthreads();
          if(threadIdx.x >= stride ){
            partial_sum_t[i] +=temp;
          }
          __syncthreads();

     }     
     
     }
    
}

void addPartialOutput(float *output, float *partial_sum){

 

    for( int i=0;i< N; i++){
        if((i/4)==0){
            output[i] +=partial_sum[0];
        }else if((i/4)==1){
            output[i] +=partial_sum[1];
        }else if((i/4)==2){
            output[i] +=partial_sum[2];
        }
        else if((i/4)==3){
            output[i] +=partial_sum[3];
        }

    }

}

void initArray(float *vector_h){

    for(int i=0;i < N; i++){
        vector_h[i]=1;
    }
}


int main(){
size_t size= N * sizeof(float);
size_t p_size= P * sizeof(float);

float *vector_h;
float *partial_sum_h;
float *output_h;
// allocate memories on the host
vector_h=(float*)malloc(size);
output_h=(float*)malloc(size);

partial_sum_h=(float*)malloc(p_size);

initArray(vector_h);


float *vector_d;
float *partial_sum_d;
float *partial_sum_d_t;

float *input_d;
hipError_t err;

// allocates memories on the device
err=hipMalloc((void**)&input_d,size);
 if(err !=hipSuccess){
               fprintf(stderr, "hipMalloc d_A error: %s\n", hipGetErrorString(err)); exit(EXIT_FAILURE); 
       }

err=hipMalloc((void**)&vector_d,size);
 if(err !=hipSuccess){
               fprintf(stderr, "hipMalloc d_A error: %s\n", hipGetErrorString(err)); exit(EXIT_FAILURE); 
       }
err = hipMalloc((void**)&partial_sum_d,p_size);
 if(err !=hipSuccess){
               fprintf(stderr, "hipMalloc d_A error: %s\n", hipGetErrorString(err)); exit(EXIT_FAILURE); 
       }

       err = hipMalloc((void**)&partial_sum_d_t,p_size);
 if(err !=hipSuccess){
               fprintf(stderr, "hipMalloc d_A error: %s\n", hipGetErrorString(err)); exit(EXIT_FAILURE); 
       }

//copy memory from host to device
    hipMemcpy(input_d,vector_h,size,hipMemcpyHostToDevice);
     
     // inilizes thread and threadblocks
     dim3 threadPerBlock(4);
     dim3 numberOfBlocks(4);


     scanKernel <<< numberOfBlocks, threadPerBlock >>> (input_d, vector_d, partial_sum_d);
     hipDeviceSynchronize();
      //scan partial sum array

    //std::cout<< blockDim.x << std::endl;


     scanPartial <<<numberOfBlocks, threadPerBlock >>> (partial_sum_d,partial_sum_d_t);
           //cudaDeviceSynchronize();


           

// copy memories fro device to host
     hipMemcpy(output_h, vector_d,size,hipMemcpyDeviceToHost);
     hipMemcpy(partial_sum_h,partial_sum_d_t,p_size,hipMemcpyDeviceToHost);
     




     // This adds the partial sum array to the previously scanned array for final result
    addPartialOutput(output_h, partial_sum_h);
    

   // output display
 for (int i=0;i<N;i++){
    std::cout<< output_h[i]<< std::endl;
   }

// free some device memories
hipFree(vector_d);
hipFree(partial_sum_d);
hipFree(partial_sum_d_t);

hipFree(input_d);

// free some host memories
delete[] vector_h;
delete[] partial_sum_h;
delete[] output_h;

    return 0;
}